#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#define PI 3.14159265358979323846

inline void checkCudaErrors(hipError_t err) //cuda Error handle Function
{    
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s.\n", hipGetErrorString(err));
		return;
	}
}

//
// __device__ double abs(double a) {
//	 if (a > 0)
//		 return a;
//	 else
//		 return -a;
//}
//
// __device__ double pow(double a, int b) {
//	 double sum = a;
//	 for (int i = 1; i < b; i++)
//		 sum *= a;
//	 if (b == 0)
//		 return 1;
//	 else
//		 return sum;
// }
//
__device__ void getW_xy_a(double *n, double p) {
	int a = -0.5;
	int pn = (int)p;  //�����������
	double u = p - pn;  //���С������
	double np[4];
	np[0] = 1 + u;   //>1
	np[1] = u;        //<1
	np[2] = 1 - u;     //<1
	np[3] = 2 - u;   //>1
	n[0] = a * pow(abs(np[0]), 3) - 5 * a * pow(abs(np[0]), 2) + 8 * a * abs(np[0]) - 4 * a;
	n[1] = (a + 2) * pow(abs(np[1]), 3) - (a + 3) * pow(abs(np[1]), 2) + 1;
	n[2] = (a + 2) * pow(abs(np[2]), 3) - (a + 3) * pow(abs(np[2]), 2) + 1;
	n[3] = a * pow(abs(np[3]), 3) - 5 * a * pow(abs(np[3]), 2) + 8 * a * abs(np[3]) - 4 * a;
};


__global__ void extendPic(double *pixel, double *bigPic, double width, double height, double factor) {

	//int y = blockDim.y * blockIdx.y + threadIdx.y;
	//int x = blockDim.x * blockIdx.x + threadIdx.x;  //��������

	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;  //��������

	double bWidth = factor * width;
	double bHeight = factor * height;

	if (x >= 2 && x < bWidth - 4 && y >= 2  && y < bHeight - 4) {
		double bI = (double)y * 1 / factor; //�Ŵ�ǰ��λ�� y
		double bJ = (double)x * 1 / factor; // x

		double Wx[4], Wy[4];  //xy��Ȩ��
		getW_xy_a(Wx, bI);
		getW_xy_a(Wy, bJ);

		double element = 0;

		for (int a = 0; a < 4; a++) {
			for (int b = 0; b < 4; b++) {// a00 * x0y0 + a10*x
				/*element += (Vec3f)(dealPic.at<cv::Vec3b>((int)bI + a - 1, (int)bJ + b - 1)) * Wx[a] * Wy[b];*/
			/*	if ((bJ + (b - 1)) >= 0 && (bJ + (b - 1)) < width  && (bI + a - 1) >= 0 && (bI + a - 1) < height)*/
					element += pixel[(int)(bI + a - 1) * (int)width + (int)(bJ + (b - 1))] * Wx[a] * Wy[b];
			}
		}

		bigPic[(int)(y * bWidth + x)] = element;

	}
	/*else
		bigPic[int(y * bWidth + x)] = 0;*/

}

__global__ void rotatePic(double *pixel, double *rPic, double width,
	double height, double rWidth, double rHeight, double angle) {

	angle = angle / 180.0f * PI;
	double cosa = cos(angle);
	double sina = sin(angle);

	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;  //��������

	//����ӳ��Ѱ��ԭͼ�е�λ��
	int Rx = round(double(y - rHeight / 2.0f)*cosa - double(x - rWidth / 2.0f)*sina + height / 2.0f);
	int Ry = round(double(y - rHeight / 2.0f)*sina + double(x - rWidth / 2.0f)*cosa + width / 2.0f);


	double element = 0; //����ֵ
	double Wx[4], Wy[4];  //xy��Ȩ��

	getW_xy_a(Wx, Rx);
	getW_xy_a(Wy, Ry);
	//���������صĴ�С
	if (x >= 0 && x < rWidth  && y >= 0 && y < rHeight ) {
		for (int a = 0; a < 4; a++) {
			for (int b = 0; b < 4; b++) {// a00 * x0y0 + a10*x
				if (Rx + a - 1 < 0 || Rx + a - 1 >= height || Ry + b - 1 < 0 || Ry + b - 1 >= width)    //��ֹԽ�� ����ԭͼ��
					rPic[y * (int)rWidth + x] = 0;
				else
				{
					element += pixel[int((Rx + a - 1) * width) + ((int)Ry + b - 1)] * Wx[a] * Wy[b];
				}
			}
		}
		rPic[y * (int)rWidth + x] = element;
	}
}

__global__ void gayFilter(double *pixel, double* wight,  double Width, double Height, int filterSize) {
	int skipSize = filterSize / 2; //�����������


	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;  //��������

	if (x >= skipSize && x < Width - 1 - filterSize && y >= skipSize && y < Height - 1 - filterSize) { //��ֹԽ��
	
		double * filterArray = (double*)malloc(sizeof(double) * filterSize * filterSize);  //�洢������ڵ�����ֵ
		int c_cout = 0;
		for (int k = skipSize, count_i = 0; count_i < filterSize; count_i++) {
			// i -1 j -1    i - 0, j - 1 , i +1, j -1 
			// 0 1 2 3 4 5 p[5]
			//�����ص��������
			for (int p = skipSize, count_j = 0; count_j < filterSize; count_j++) {
				filterArray[c_cout] = pixel[ (y - k) * (int)Width + x - p];
				p--;
				c_cout++;
			}
			k--;
		}
		double value = 0;
		//��Ȩ���
		for (int i = 0; i < filterSize* filterSize; i++)
			value += filterArray[i] * wight[i];

		pixel[y * (int)Width + x] = value;
		free(filterArray);

	}

}


extern "C" void  BicubicInterpolation_host(double *pixel, double *bigPic, double Width, double Height, double factor) {
	double * oPic;
	double * bPic;
	//double * rPic;

	checkCudaErrors(hipMalloc((void**)&oPic, sizeof(double) * Width * Height));
	checkCudaErrors(hipMalloc((void**)&bPic, sizeof(double) * Width * Height * factor * factor));
	//checkCudaErrors(hipMalloc((void**)&rPic, (sizeof(double) * Width * Height * factor * factor)));
	checkCudaErrors(hipMemcpy(oPic, pixel, sizeof(double) * Width * Height, hipMemcpyHostToDevice));

	int length = 12;
	dim3 dimGrid(Width * factor / length, Height * factor / length);
	/*dim3 dimGrid((Width * factor + dimBlock.x - 1) / dimBlock.x, (Height * factor + dimBlock.y -
		1) / dimBlock.y);*/
	dim3 dimBlock(length, length);
	extendPic << <dimGrid, dimBlock >> > (oPic, bPic, Width, Height, factor);

	checkCudaErrors(hipMemcpy(bigPic, bPic, sizeof(double) * Width * Height * factor * factor, hipMemcpyDeviceToHost));

	hipFree(oPic);
	hipFree(bPic);
	//hipFree(rPic);
}

extern "C" void RotatePictureByBic_host(double *pixel, double *rPic, double Width, 
	double Height, double rWidth, double rHeight, double angle) {
	double * oPic;
	double * bPic;


	checkCudaErrors(hipMalloc((void**)&oPic, sizeof(double) * Width * Height));  //ԭ��ͼƬ��С
	checkCudaErrors(hipMalloc((void**)&bPic, sizeof(double) * rWidth * rHeight));//��תͼƬ��С

	checkCudaErrors(hipMemcpy(oPic, pixel, sizeof(double) * Width * Height, hipMemcpyHostToDevice));//��ֵ����cup��gpu
	int length = 32;
	dim3 dimGrid(length , length ); //�ڴ������С
	///*dim3 dimGrid((Width * factor + dimBlock.x - 1) / dimBlock.x, (Height * factor + dimBlock.y -
	//	1) / dimBlock.y);*/
	dim3 dimBlock(length,length );
	rotatePic << <dimGrid, dimBlock >> > (oPic, bPic, Width, Height, rWidth, rHeight,  angle);

	checkCudaErrors(hipMemcpy(rPic, bPic, sizeof(double) * rWidth * rHeight, hipMemcpyDeviceToHost));

	hipFree(oPic);
	hipFree(bPic);
}


extern "C" void GayssianFilter_host(double *pixel,double* wight, double Width, double Height, int filterSize) {
	double * oPic;
	double * Wight;
	double * filterArray;

	checkCudaErrors(hipMalloc((void**)&oPic, sizeof(double) * Width * Height));  //ԭ��ͼƬ��С
	checkCudaErrors(hipMalloc((void**)&Wight, sizeof(double) * filterSize * filterSize));//Ȩ�ؾ����С
	checkCudaErrors(hipMalloc((void**)&filterArray, sizeof(double) * filterSize * filterSize));//���ھ����С

	checkCudaErrors(hipMemcpy(oPic, pixel, sizeof(double) * Width * Height, hipMemcpyHostToDevice));//��ֵ����cup��gpu
	checkCudaErrors(hipMemcpy(Wight, wight, sizeof(double) * filterSize * filterSize, hipMemcpyHostToDevice));//��ֵ����cup��gpu
	int length = 32;
	dim3 dimGrid(length, length); //�ڴ������С
	///*dim3 dimGrid((Width * factor + dimBlock.x - 1) / dimBlock.x, (Height * factor + dimBlock.y -
	//	1) / dimBlock.y);*/
	dim3 dimBlock(length, length);
	//
	gayFilter<<<dimGrid, dimBlock >> > (oPic, Wight,  Width, Height, filterSize);

	checkCudaErrors(hipMemcpy(pixel , oPic, sizeof(double) * Width * Height, hipMemcpyDeviceToHost));

	hipFree(oPic);
	hipFree(Wight);
	hipFree(filterArray);
}